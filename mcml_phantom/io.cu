#include "header.h"

void output_fiber(SimulationStruct* sim, float *data, char* output)
{
	ofstream myfile;
	// myfile.open ("GPUMC_output.txt",ios::app); //Wang modified for fitting filename
	myfile.open(output, ios::app);

	double scale1 = (double)0xFFFFFFFF*(double)sim->number_of_photons;
	if(NORMAL)
	{
		for(int i = 0; i < NUM_OF_DETECTOR; i++)
		{
			myfile << double(data[i]/scale1)  << "\t";
		}
	}
	else
	{
		for(int i = 0; i < NUM_OF_DETECTOR; i++)
		{
			myfile << double(data[i]/scale1)  << " ";
		}
	}
	myfile << endl;
    myfile.close();
}


int read_mua_mus(SimulationStruct** simulations, char* input)
{
	// parameters to be modified
	unsigned long number_of_photons = NUMBER_PHOTONS ;  
	const int n_simulations = NUMBER_SIMULATION;
	
	int n_layers = 1;                                   // Wang modified; double layer, default value = 2
	//float medium_n = 1.0;								// float medium_n = 1.33;   // refractive index of medium // YU-modified
	float medium_n = 1.457;  //Wang-modified
	// float lower_thickness = 10.0;						// YU-modified
	float tissue_n = 1.4;                            // refractive index of tissue
	float g_factor = 0.0;                            // anisotropic					//YU-modified

	float start_weight;
	float upper_thickness = 20;

	
	// read the file 
	fstream myfile;
	// myfile.open("GPUMC_input.txt"); //Wang modified for fitting filename
	myfile.open(input);  //Wang modified

	float up_mua[n_simulations];
	float up_mus[n_simulations];


	for (int i = 0; i < n_simulations; i++){
		myfile >> up_mua[i] >> up_mus[i];
		
	}
	myfile.close();


	// Allocate memory for the SimulationStruct array
	*simulations = (SimulationStruct*) malloc(sizeof(SimulationStruct)*n_simulations);
	if(*simulations == NULL){perror("Failed to malloc simulations.\n");return 0;}//{printf("Failed to malloc simulations.\n");return 0;}

	for(int i = 0;i < n_simulations; i++)
	{
		(*simulations)[i].number_of_photons=number_of_photons;
		(*simulations)[i].n_layers = n_layers;

		// Allocate memory for the layers (including one for the upper and one for the lower)
		(*simulations)[i].layers = (LayerStruct*) malloc(sizeof(LayerStruct)*(n_layers+2));
		if((*simulations)[i].layers == NULL){perror("Failed to malloc layers.\n");return 0;}//{printf("Failed to malloc simulations.\n");return 0;}

		// Set upper refractive index (medium)
		(*simulations)[i].layers[0].n = medium_n;	//(*simulations)[i].layers[0].n = medium_n[i]; //YU-modified

		// Set the parameters of tissue (upper layer)
		(*simulations)[i].layers[1].n     = tissue_n;
		(*simulations)[i].layers[1].mua   = up_mua[i];
		(*simulations)[i].layers[1].g = g_factor;			//(*simulations)[i].layers[1].g     = g_factor; //YU-modified 
		(*simulations)[i].layers[1].z_min = 0;
		(*simulations)[i].layers[1].z_max = upper_thickness;	//(*simulations)[i].layers[1].z_max = upper_thickness; //YU-modified
		(*simulations)[i].layers[1].mutr  = 1.0f/(up_mua[i]+up_mus[i]);



		// Set lower refractive index (medium)
		(*simulations)[i].layers[n_layers + 1].n = medium_n;		//(*simulations)[i].layers[n_layers+1].n = medium_n[i]; //YU-modified

		//calculate start_weight
		double n1=(*simulations)[i].layers[0].n;
		double n2=(*simulations)[i].layers[1].n;
		double r = (n1-n2)/(n1+n2);
		r = r*r;
		start_weight = (unsigned int)((double)0xffffffff*(1-r));  
		//start_weight = 1-r;  
		//printf("Start weight=%u\n",start_weight);
		(*simulations)[i].start_weight=start_weight;
	}
	return n_simulations;
}